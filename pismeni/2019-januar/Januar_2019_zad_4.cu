#include "hip/hip_runtime.h"
% % cu
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define N 9
#define BLOCKSIZE 4

        __global__ void
        kernel(float *a, float *b)
{
  __shared__ float arrayPart[BLOCKSIZE + 2];
  int mySharedId = blockDim.x * blockIdx.x + threadIdx.x;
  if (mySharedId < (N + 2))
  {
    arrayPart[threadIdx.x] = a[mySharedId];
    if (threadIdx.x >= BLOCKSIZE - 2 && mySharedId + 2 < (N + 2))
      arrayPart[threadIdx.x + 2] = a[mySharedId + 2];
  }

  __syncthreads();

  if (mySharedId < N)
    b[mySharedId] = (3 * arrayPart[threadIdx.x] + 10 * arrayPart[threadIdx.x + 1] + 7 * arrayPart[threadIdx.x + 2]) / 20.f;
}

int main(void)
{
  float A[N + 2], B[N];

  float *device_a, *device_b;
  hipMalloc((void **)&device_a, (N + 2) * sizeof(float));
  hipMalloc((void **)&device_b, N * sizeof(float));
  for (int i = 0; i < N + 2; i++)
  {
    A[i] = i + 2;
    printf("%f ", A[i]);
  }

  hipMemcpy(device_a, A, (N + 2) * sizeof(int), hipMemcpyHostToDevice);

  int blockSize = BLOCKSIZE;
  int gridSize = (int)ceil(((float)N) / blockSize);
  printf("\ngridSize: %d\n", gridSize);

  kernel<<<gridSize, blockSize>>>(device_a, device_b);
  hipMemcpy(B, device_b, N * sizeof(float), hipMemcpyDeviceToHost);

  printf("A: ");
  for (int i = 0; i < N + 2; i++)
    printf("%f ", A[i]);
  printf("\n\nB: ");
  for (int i = 0; i < N; i++)
    printf("%f ", B[i]);
  printf("\n\n");
}